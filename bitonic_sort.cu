
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <ctime>

using namespace std;

#define THREADS 8
#define BLOCKS  8
#define n THREADS * BLOCKS

int *d_arr, *h_arr;

__global__ void bitonicsort(int *d_arr, int j, int k) {
    unsigned int idx, ixj;
    idx = threadIdx.x + blockDim.x * blockIdx.x;
    ixj = idx ^ j;

    if (ixj > idx) {
        if ((idx&k) == 0) {
            if (d_arr[idx] > d_arr[ixj]) {
                int temp = d_arr[idx];
                d_arr[idx] = d_arr[ixj];
                d_arr[ixj] = temp;
            }
        }
        if ((idx&k) != 0) {
            if (d_arr[idx] < d_arr[ixj]) {
                int temp = d_arr[idx];
                d_arr[idx] = d_arr[ixj];
                d_arr[ixj] = temp;
            }
        }
    }
}

void runBitonicsort(int *h_arr, size_t N) {
    dim3 blocks(BLOCKS,1);
    dim3 threads(THREADS,1);
    
    cout << "Unsorted Array: ";
    for (int i = 0; i < n; i++)
        cout << h_arr[i] << " ";
    cout <<  endl;
   
    hipMalloc((void **) &d_arr, N);
   
    hipMemcpy(d_arr, h_arr, N, hipMemcpyHostToDevice);
  
    for (int k = 2; k <= n; k <<= 1) {                      //Move k to the left by 1 bit, replacing it
        for (int j = k >> 1; j > 0; j--) {                  //Move k to the right by 1 bit, reducing j
            bitonicsort<<<blocks, threads>>>(d_arr, j, k);
        }
    }
    hipDeviceSynchronize();
  
    hipMemcpy(h_arr, d_arr, N, hipMemcpyDeviceToHost);
  
    cout << "Sorted Array: ";
    for (int i = 0; i < n; i++)
        cout << h_arr[i] << " ";
    cout <<  endl;
    
    hipFree(d_arr);
    free(h_arr);
}

int main(void) {
    srand(time(0));
    size_t N = n * sizeof(int);
    h_arr = (int*) malloc(N);
    
    cout << "Filling array with random numbers..." << endl;
    for (int i = 0; i < n; i++) 
        h_arr[i] = rand() % 100;

    runBitonicsort(h_arr, N);
    return 0;
}
