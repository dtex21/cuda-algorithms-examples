#include "hip/hip_runtime.h"
//For comments go to cash_karp.cu
#include <stdio.h>

using namespace std;

double *x, *y, *d_x, *d_y;
const double step = 0.1;

__device__ double f(double x, double y) {
    return 3 * x * x + y;
}

__global__ void runge_kutta(double *x, double *y) {
    double k1, k2, k3, k4;
    int x_max = 0;
    int idx = threadIdx.x;
    
    while (x_max < 10) {    
        printf("X: %f\tY: %f\n", *x, *y); 
        
        k1 = step * f(x[idx], y[idx]);
        k2 = step * f((x[idx] + step/2), (y[idx] + k1/2));
        k3 = step * f((x[idx] + step/2), (y[idx] + k2/2));
        k4 = step * f((x[idx] + step), (y[idx] + k3));
        
        y[idx] += (k1 + 2*k2 + 2*k3 + k4) / 6;
        x[idx] += step;
        x_max++;
    }
}

__global__ void parent_runge_kutta(double *x, double *y) {
    int i = 0;
    
    printf("!---- Start of Process ----!\n");
        
    while (i < 2) {
        runge_kutta <<< 1, 1 >>>(x, y);
        i++;
        hipDeviceSynchronize();
    }
    
    printf("!----- End of Process -----!\n");
}

int main() {
    size_t N = sizeof(double);

    hipMalloc((void **) &d_x, N);
    hipMalloc((void **) &d_y, N);
    
    x = (double *)malloc(N);
    y = (double *)malloc(N);
    
    hipMemcpy(d_x, x, N, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N, hipMemcpyHostToDevice);
    
    parent_runge_kutta <<< 1, 1 >>>(d_x, d_y);
    
    hipMemcpy(x, d_x, N, hipMemcpyDeviceToHost);
    hipMemcpy(y, d_y, N, hipMemcpyDeviceToHost);
    
    printf("Final Result = X: %g, Y: %.4g\n", *x, *y);
    
    free(x);    free(y);
    hipFree(d_x);  hipFree(d_y);
    return 0;
}
